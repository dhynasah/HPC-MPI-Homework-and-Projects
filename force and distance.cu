/*
 * Dhynasah Cakir 
 * force and distance 
 * homework 7 
 * 10/17/18
 */

#include <stdio.h>
#include <stdlib.h>

// global directive for kernel 
_global_ void cu_fillArray( int *force_d, int *dist_d, int *work_d) 
{
	int x;  
	// using built in variables 
	x = blockIdx.x * BLOCK_SIZE + threadId.x; 
	force_d[x] = rand()%100+1; 
	dist_d[x] = rand()%10+1; 
	work_d[x] = force[x] * dist_d[x]; 

}

extern "C" void fillArray (int *force, int *dist, int *work, int arraySize)
{
	// force_d, dist_d, and work_d are the GPU counterparts of the arrays that exists in host memory 
	int *force_d;
	int *dist_d;
	int *work_d;
	hipError_t result;

	// allocate space in the device 
	result = hipMalloc ((void**) &force_d, sizeof(int) * arraySize);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMalloc (force) failed.");
		exit(1);
	}
	result = hipMalloc ((void**) &dist_d, sizeof(int) * arraySize);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMalloc (dist) failed.");
		exit(1);
	}
	result = hipMalloc((void**) &work_d, sizeof(int) * arraySize);
	if( result != hipSuccess) {
		fprintf(stderr, "hipMalloc (work) failed.");
	}

	//copy the arrays from host to the device 
	result = hipMemcpy (force_d, force, sizeof(int) * arraySize, hipMemcpyHostToDevice);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy host->dev (force) failed.");
		exit(1);
	}
	result = hipMemcpy (dist_d, dist, sizeof(int) * arraySize, hipMemcpyHostToDevice);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy host->dev (dist) failed.");
		exit(1);
	}
	
	result = hipMemcpy (work_d, work, sizeof(int) * arraySize, hipMemcpyHostToDevice);
	if( result != hipSuccess){
		fprintf(stderr, "hipMalloc (work) failed.");
		exit(1); 
	}

	// set execution configuration
	dim3 dimblock (BLOCK_SIZE);
	dim3 dimgrid (arraySize/BLOCK_SIZE);

	// actual computation: Call the kernel
	cu_fillArray <<<dimgrid, dimblock>>> (force_d, dist_d, work_d);

	// transfer results back to host
	result = hipMemcpy (work, work_d, sizeof(int) * arraySize, hipMemcpyDeviceToHost);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy host <- dev (work) failed.");
		exit(1);
	}
	result = hipMemcpy (dist, dist_d, sizeof(int) * arraySize, hipMemcpyDeviceToHost);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy host <- dev (dist) failed.");
		exit(1);
	}
	result = hipMemcpy (force, force_d, sizeof(int) * arraySize, hipMemcpyDeviceToHost);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy host <- dev (force) failed.");
		exit(1);
	}
	
	// release the memory on the GPU 
	result = hipFree (force_d);
	if (result != hipSuccess) {
		fprintf(stderr, "hipFree (block) failed.");
		exit(1);
	}
	result = hipFree (dist_d);
	if (result != hipSuccess) {
		fprintf(stderr, "hipFree (thread) failed.");
		exit(1);
	}
	result = hipFree (work_d);
	if (result != hipSuccess) {
		fprintf(stderr, "hipFree (thread) failed.");
		exit(1);
	}
}


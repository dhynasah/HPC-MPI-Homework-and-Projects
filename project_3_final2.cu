/*
 * project_3.cu 
 * includes setup function called from driver program 
 * includes kernel function 'cu_claculateDiffusion()'
 */
 

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

 
#define BLOCK_SIZE 256

 

__global__ void updateDensity(double *newDensity, double *oldDensity, int SIZEOFARRAY){
	int index; 
	index = blockIdx.x *BLOCK_SIZE + threadIdx.x; 
	if(index == 0){
		newDensity[index] = (oldDensity[index]*2+oldDensity[index+1])/3;
	} else if(index == SIZEOFARRAY) {
		newDensity[index] = (oldDensity[index-1]+oldDensity[index]*2)/3;
	} else {
		newDensity[index] = (oldDensity[index-1]+oldDensity[index]+oldDensity[index+1])/3;
	}	
	
}


extern "C" void simulate(double *Density1, double* Density2, int SIZEOFARRAY, int TimeSteps)
{
	double *Density1_d; 
	double *Density2_d; 
	hipError_t result;
	 
	//allocate space in the device 
	result = hipMalloc ((void**) &Density1_d, sizeof(double) * SIZEOFARRAY);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMalloc (Density1) failed.");
		exit(1);
	}
	result = hipMalloc ((void**) &Density2_d, sizeof(double) * SIZEOFARRAY);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMalloc (Density2) failed.");
		exit(1);
	}

	//copy the arrays from host to the device 
	result = hipMemcpy (Density1_d, Density1 , sizeof(double) * SIZEOFARRAY, hipMemcpyHostToDevice);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy host->dev (Density1) failed.");
		exit(1);
	}
	result = hipMemcpy (Density2_d, Density2, sizeof(double) * SIZEOFARRAY, hipMemcpyHostToDevice);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy host->dev (Density2) failed.");
		exit(1);
	}
	
	//set exectuion configuration 
	dim3 dimblock (BLOCK_SIZE);
	dim3 dimgrid (SIZEOFARRAY/BLOCK_SIZE); 
	//function that calls the GPU
	 int i;
	for (i=1; i<= TimeSteps; i++){
		if (i%2 == 0) {
            updateDensity<<<dimgrid,dimblock>>>(Density1_d, Density2_d, SIZEOFARRAY); 
		 
        } 
	else {
              updateDensity<<<dimgrid,dimblock>>>(Density2_d, Density1_d, SIZEOFARRAY);		
        }
		
	}
	
	
	result = hipMemcpy (Density1, Density1_d, sizeof(double) * SIZEOFARRAY, hipMemcpyDeviceToHost);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy host <- dev (Density1) failed.");
		exit(1);
	}
	result = hipMemcpy (Density2, Density2_d, sizeof(double) * SIZEOFARRAY, hipMemcpyDeviceToHost);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy host <- dev (Density2) failed.");
		exit(1);
	}
	
	// release the memory on the GPU 
	result = hipFree (Density1_d);
	if (result != hipSuccess) {
		fprintf(stderr, "hipFree (Density1) failed.");
		exit(1);
	}
	result = hipFree (Density2_d);
	if (result != hipSuccess) {
		fprintf(stderr, "hipFree (Density2) failed.");
		exit(1);
	}
	}